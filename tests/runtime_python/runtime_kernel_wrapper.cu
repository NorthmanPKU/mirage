#include "hip/hip_runtime.h"
#include "argmax.cuh"
#include "linear.cuh"
#include "norm_linear.cuh"
#include "silu_mul_linear.cuh"
#include "single_batch_decoding.cuh"
#include "single_batch_gqa.cuh"
#include "embedding.cuh"
#include "prompt_lookup.cuh"
#include "bfloat16.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

// using kernel::argmax_kernel;
using kernel::linear_kernel;
using kernel::norm_linear_task_impl;
using kernel::silu_mul_linear_task_impl;
using kernel::single_batch_decoding_kernel;
using kernel::single_batch_gqa_kernel;
using kernel::embedding_kernel;
using kernel::find_ngram_partial_kernel;
using kernel::find_ngram_global_kernel;
using bfloat16 = type::bfloat16_t;

template <typename T>
__global__ void single_batch_gqa_kernel_wrapper(void const *qkv_ptr,
                                                void *k_cache_ptr,
                                                void *v_cache_ptr,
                                                void *output_ptr,
                                                size_t seq_len,
                                                bool qk_norm,
                                                bool rotary_embed,
                                                void const *qnorm_weight_ptr,
                                                void const *knorm_weight_ptr,
                                                void const *cos_ptr,
                                                void const *sin_ptr,
                                                float q_eps,
                                                float k_eps) {
  single_batch_gqa_kernel<T, 4>(qkv_ptr,
                                k_cache_ptr,
                                v_cache_ptr,
                                output_ptr,
                                seq_len,
                                qk_norm,
                                rotary_embed,
                                qnorm_weight_ptr,
                                knorm_weight_ptr,
                                cos_ptr,
                                sin_ptr,
                                q_eps,
                                k_eps);
}

void single_batch_gqa(
    torch::Tensor qkv,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor output,
    size_t seq_len,
    bool qk_norm,
    bool rotary_embed,
    torch::optional<torch::Tensor> qnorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> knorm_weight = torch::nullopt,
    torch::optional<torch::Tensor> cos = torch::nullopt,
    torch::optional<torch::Tensor> sin = torch::nullopt,
    float q_eps = 0.0f,
    float k_eps = 0.0f) {
  void const *qkv_ptr = qkv.data_ptr();
  void *k_cache_ptr = k_cache.data_ptr();
  void *v_cache_ptr = v_cache.data_ptr();
  void *output_ptr = output.data_ptr();

  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 88888;

  void const *qnorm_weight_ptr = qk_norm ? qnorm_weight->data_ptr() : nullptr;
  void const *knorm_weight_ptr = qk_norm ? knorm_weight->data_ptr() : nullptr;
  void const *cos_ptr = rotary_embed ? cos->data_ptr() : nullptr;
  void const *sin_ptr = rotary_embed ? sin->data_ptr() : nullptr;

  hipFuncSetAttribute(reinterpret_cast<const void*>(single_batch_gqa_kernel_wrapper<bfloat16>),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  single_batch_gqa_kernel_wrapper<bfloat16>
      <<<grid_dim, block_dim, smem_size>>>(qkv_ptr,
                                           k_cache_ptr,
                                           v_cache_ptr,
                                           output_ptr,
                                           seq_len,
                                           qk_norm,
                                           rotary_embed,
                                           qnorm_weight_ptr,
                                           knorm_weight_ptr,
                                           cos_ptr,
                                           sin_ptr,
                                           q_eps,
                                           k_eps);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// RMSNorm Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void norm_linear_kernel_wrapper(void const *input_ptr,
                                           void const *norm_weight_ptr,
                                           void const *weight_ptr,
                                           float eps,
                                           void *output_ptr) {
  norm_linear_task_impl<T,
                        BATCH_SIZE,
                        OUTPUT_SIZE,
                        REDUCTION_SIZE,
                        OUTPUT_SIZE>(
      input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_norm_linear(void const *input_ptr,
                        void const *norm_weight_ptr,
                        void const *weight_ptr,
                        float eps,
                        void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      norm_linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  norm_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, norm_weight_ptr, weight_ptr, eps, output_ptr);
}

void norm_linear(torch::Tensor input,
                 torch::Tensor norm_weight,
                 torch::Tensor weight,
                 float eps,
                 torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *norm_weight_ptr = norm_weight.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_norm_linear<bfloat16, 1, 16, 4096>(input_ptr, norm_weight_ptr,
                                                weight_ptr, eps, output_ptr);
      break;
    case 32:
      launch_norm_linear<bfloat16, 1, 32, 4096>(input_ptr, norm_weight_ptr,
                                                weight_ptr, eps, output_ptr);
      break;
    case 64:
      launch_norm_linear<bfloat16, 1, 64, 4096>(input_ptr, norm_weight_ptr,
                                                weight_ptr, eps, output_ptr);
      break;
    case 256:
      launch_norm_linear<bfloat16, 1, 256, 4096>(input_ptr, norm_weight_ptr,
                                                weight_ptr, eps, output_ptr);
      break;
    case 1600:
      launch_norm_linear<bfloat16, 1, 1600, 4096>(input_ptr, norm_weight_ptr,
                                                weight_ptr, eps, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// SiLU MUL Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void silu_mul_linear_kernel_wrapper(void const *input_ptr,
                                               void const *weight_ptr,
                                               void const *bias_ptr,
                                               void *output_ptr) {
  silu_mul_linear_task_impl<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, bias_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_silu_mul_linear(void const *input_ptr,
                            void const *weight_ptr,
                            void const *bias_ptr,
                            void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(silu_mul_linear_kernel_wrapper<T),
                                                      BATCH_SIZE,
                                                      OUTPUT_SIZE,
                                                      REDUCTION_SIZE>,
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       smem_size);

  silu_mul_linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, bias_ptr, output_ptr);
}

void silu_mul_linear(torch::Tensor input,
                     torch::Tensor weight,
                     torch::Tensor bias,
                     torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *bias_ptr = bias.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_silu_mul_linear<bfloat16, 1, 16, 12288>(input_ptr, weight_ptr,
                                                     bias_ptr, output_ptr);
      break;
    case 32:
      launch_silu_mul_linear<bfloat16, 1, 32, 12288>(input_ptr, weight_ptr,
                                                     bias_ptr, output_ptr);
      break;
    case 64:
      launch_silu_mul_linear<bfloat16, 1, 64, 12288>(input_ptr, weight_ptr,
                                                     bias_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Linear

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
__global__ void linear_kernel_wrapper(void const *input_ptr,
                                      void const *weight_ptr,
                                      void const *residual_ptr,
                                      void *output_ptr) {
  linear_kernel<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>(
      input_ptr, weight_ptr, residual_ptr, output_ptr);
}

template <typename T, int BATCH_SIZE, int OUTPUT_SIZE, int REDUCTION_SIZE>
void launch_linear(void const *input_ptr,
                   void const *weight_ptr,
                   void const *residual_ptr,
                   void *output_ptr) {
  dim3 grid_dim(1, 1, 1);
  dim3 block_dim(128, 1, 1);
  size_t smem_size = 112640;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      linear_kernel_wrapper<T), BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      smem_size);

  linear_kernel_wrapper<T, BATCH_SIZE, OUTPUT_SIZE, REDUCTION_SIZE>
      <<<grid_dim, block_dim, smem_size>>>(
          input_ptr, weight_ptr, residual_ptr, output_ptr);
}

void linear(torch::Tensor input,
            torch::Tensor weight,
            torch::Tensor residual,
            torch::Tensor output) {

  void const *input_ptr = input.data_ptr();
  void const *weight_ptr = weight.data_ptr();
  void const *residual_ptr = residual.data_ptr();
  void *output_ptr = output.data_ptr();

  switch (output.size(1)) {
    case 16:
      launch_linear<bfloat16, 1, 16, 4096>(input_ptr, weight_ptr,
                                           residual_ptr, output_ptr);
      break;
    case 32:
      launch_linear<bfloat16, 1, 32, 4096>(input_ptr, weight_ptr,
                                           residual_ptr, output_ptr);
      break;
    case 64:
      launch_linear<bfloat16, 1, 64, 4096>(input_ptr, weight_ptr,
                                           residual_ptr, output_ptr);
      break;
    default:
      printf("Unsupported output size in test: %zu\n", output.size(1));
      break;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
  }
}

// Argmax

// template <typename T>
// __global__ void argmax_kernel_wrapper(void const *input_ptr, void
// *output_ptr) {
//   argmax_kernel<T, 1, 32768>(input_ptr, output_ptr);
// }

// template <typename T>
// void launch_argmax(void const *input_ptr, void *output_ptr) {
//   dim3 grid_dim(1, 1, 1);
//   dim3 block_dim(128, 1, 1);
//   size_t smem_size = 36666;

//   hipFuncSetAttribute(reinterpret_cast<const void*>(argmax_kernel_wrapper<T>),
//                        hipFuncAttributeMaxDynamicSharedMemorySize,
//                        smem_size);

//   argmax_kernel_wrapper<T>
//       <<<grid_dim, block_dim, smem_size>>>(input_ptr, output_ptr);
// }

// void argmax(torch::Tensor input, torch::Tensor output) {

//   void const *input_ptr = input.data_ptr();
//   void *output_ptr = output.data_ptr();

//   launch_argmax<bfloat16>(input_ptr, output_ptr);

//   hipError_t err = hipDeviceSynchronize();
//   if (err != hipSuccess) {
//     printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
//   }
// }

// Embedding Kernel
template <typename T, int CHUNK_SIZE, int OUTPUT_DIM_SIZE>
__global__ void embedding_kernel_wrapper(void const *input_ptr,
                                         void const *embedding_ptr,
                                         void *output_ptr) {
  int input_offset = blockIdx.x;
  int64_t const *__restrict__ input = static_cast<int64_t const *>(input_ptr) + input_offset;
  int embedding_offset = blockIdx.y * CHUNK_SIZE;
  T const *__restrict__ embedding = static_cast<T const *>(embedding_ptr) + embedding_offset;
  int output_offset = blockIdx.y * CHUNK_SIZE + blockIdx.x * OUTPUT_DIM_SIZE;
  T *__restrict__ output = static_cast<T *>(output_ptr) + output_offset;

  if (blockIdx.x == 1 && blockIdx.y == 1 && threadIdx.x == 0) {
    printf("input_offset: %d, embedding_offset: %d, output_offset: %d\n", input_offset, embedding_offset, output_offset);
  }
  embedding_kernel<T, CHUNK_SIZE, OUTPUT_DIM_SIZE>(
      input, embedding, output);
  // if (blockIdx.x == 1 && blockIdx.y == 1) {
  //   printf("input: %d, embedding: %d, output: %d\n", input, embedding, output);
  // }
}

void embedding(torch::Tensor input,
               torch::Tensor weight,
               torch::Tensor output) {

  dim3 grid_dim(input.size(1), output.size(1) / 128, 1);
  printf("grid_dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
  dim3 block_dim(128, 1, 1);

  embedding_kernel_wrapper<float, 128, 4096><<<grid_dim, block_dim>>>(
      input.data_ptr(),
      weight.data_ptr(),
      output.data_ptr());

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error in embedding: %s\n",
           hipGetErrorString(err));
  }
}

// Prompt Lookup Kernel
template <int NGRAM_SIZE, int NUM_WORKERS>
__global__ void find_ngram_partial_kernel_wrapper(long long const *__restrict__ input_ptr,
                                                  long long *__restrict__ output_id_ptr,
                                                  int input_token_num) {
  // Each block gets a pointer to its unique output slot.
  long long *block_output_ptr = output_id_ptr + blockIdx.x;
  find_ngram_partial_kernel<NGRAM_SIZE, NUM_WORKERS>(input_ptr, block_output_ptr, input_token_num);
}

template <int NGRAM_SIZE, int SPEC_LENGTH, int NUM_PARTIAL_TASKS>
__global__ void find_ngram_global_kernel_wrapper(long long const *__restrict__ input_array,
                                                 long long const *__restrict__ tokens_ptr,
                                                 long long *__restrict__ output_result,
                                                 int step) {
  find_ngram_global_kernel<NGRAM_SIZE, SPEC_LENGTH, NUM_PARTIAL_TASKS>(input_array, tokens_ptr, output_result, step);
}

void prompt_lookup(torch::Tensor all_tokens,
                   int prompt_len,
                   int ngram_size,
                   int spec_length,
                   torch::Tensor final_output) {
  
  constexpr int NUM_WORKERS = 96; // Corresponds to grid size
  dim3 partial_grid_dim(NUM_WORKERS, 1, 1);
  dim3 partial_block_dim(128, 1, 1);
  
  auto partial_output_options = torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);
  torch::Tensor partial_output = torch::full({NUM_WORKERS}, INT_MAX, partial_output_options);
  
  if (ngram_size == 3) {
    find_ngram_partial_kernel_wrapper<3, NUM_WORKERS><<<partial_grid_dim, partial_block_dim>>>(
        static_cast<long long const *>(all_tokens.data_ptr()),
        static_cast<long long *>(partial_output.data_ptr()),
        prompt_len);
  } else {
    throw std::runtime_error("Unsupported ngram_size for prompt_lookup test");
  }

  dim3 global_grid_dim(1, 1, 1);
  dim3 global_block_dim(128, 1, 1);

  if (ngram_size == 3 && spec_length == 5) {
     find_ngram_global_kernel_wrapper<3, 5, NUM_WORKERS><<<global_grid_dim, global_block_dim>>>(
        static_cast<long long const *>(partial_output.data_ptr()),
        static_cast<long long const *>(all_tokens.data_ptr()),
        static_cast<long long *>(final_output.data_ptr()),
        prompt_len);
  } else {
     throw std::runtime_error("Unsupported ngram_size/spec_length for prompt_lookup test");
  }
  
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error in prompt_lookup: %s\n",
           hipGetErrorString(err));
  }
}

// pybind11 bindings

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("prompt_lookup", &prompt_lookup, "Prompt lookup kernel");
  m.def("embedding", &embedding, "Embedding kernel");
  m.def("linear", &linear, "Linear kernel");
  // m.def("argmax", &argmax, "argmax kernel");
  m.def("norm_linear", &norm_linear, "RMSNorm Linear kernel");
  m.def("silu_mul_linear", &silu_mul_linear, "SILU MUL Linear kernel");
  // m.def("single_batch_gqa", &single_batch_gqa, "Decoding kernel");
  m.def("single_batch_gqa",
        &single_batch_gqa,
        py::arg("qkv"),
        py::arg("k_cache"),
        py::arg("v_cache"),
        py::arg("output"),
        py::arg("seq_len"),
        py::arg("qk_norm"),
        py::arg("rotary_embed"),
        py::arg("qnorm_weight") = py::none(),
        py::arg("knorm_weight") = py::none(),
        py::arg("cos") = py::none(),
        py::arg("sin") = py::none(),
        py::arg("q_eps") = 0.0f,
        py::arg("k_eps") = 0.0f);
  // m.def("single_batch_decoding",
  //       &single_batch_decoding,
  //       py::arg("qkv"),
  //       py::arg("k_cache"),
  //       py::arg("v_cache"),
  //       py::arg("output"),
  //       py::arg("seq_len"),
  //       py::arg("qk_norm"),
  //       py::arg("rotary_embed"),
  //       py::arg("qnorm_weight") = py::none(),
  //       py::arg("knorm_weight") = py::none(),
  //       py::arg("cos") = py::none(),
  //       py::arg("sin") = py::none(),
  //       py::arg("q_eps") = 0.0f,
  //       py::arg("k_eps") = 0.0f);
  // m.def("single_batch_decoding",
  //       &single_batch_decoding,
  //       "FlashAttention Decoding kernel");
}